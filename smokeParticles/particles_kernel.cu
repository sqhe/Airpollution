#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* 
 * CUDA Device code for particle simulation.
 */

#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"
#define EPSILON 0.000001
texture<float4, 3, hipReadModeElementType> noiseTex;
texture<float4, 1, hipReadModeElementType> air_ver_Tex;
texture<float4, 1, hipReadModeElementType> air_wind_Tex;
texture<int4, 1, hipReadModeElementType> air_tet_Tex;
texture<int4, 1, hipReadModeElementType> air_neighbor_Tex;
texture<float4, 1, hipReadModeElementType> start_pos_Tex;
texture<int1, 1, hipReadModeElementType> start_tetId_Tex;
float4* d_start_pos;
int1 * d_start_tetId;
// simulation parameters
__constant__ SimParams params;
__constant__ float d_activeArea[6];
//get ray triangle intersection point
__device__
bool RayTriangleIntersect( float3 v0,   
						   float3 v1,
						   float3 v2,
						   float3 rayStart,
						   float3 rayDir,   
                           float&   t,
						   float previous_t)   
{   
    //Does ray parallel with triangle?   
    float3 e1, e2, p, s, q;   
    float u, v, tmp;   
    e1 = v1 - v0;   
    e2 = v2 - v0;   
    p=cross(rayDir, e2); 
    tmp = dot(p,e1);
//     if(tmp > -EPSILON && tmp < EPSILON)   
//     {   
// 		t=-1;//debug
//         return false;   
//     }   
    tmp = __fdividef(1.0f, tmp); 
    s = rayStart - v0;   
    u = dot(s, p) * tmp; 
    if(u < 0.0 || u > 1.0) 
	{
		t=-2;
        return false;   
	}
    q = cross(s, e1); 
    v = dot(rayDir, q) * tmp;  
    float uv = u+v; 
    if(v < 0.0||uv > 1.0)  
	{
		t=-3;
        return false;     
	}
    t = dot(e2, q) * tmp; 
    if(t-previous_t <= 1e-7)  
	//if(t<=previous_t)
        return false;   
    return true;   
}
//get barycentric coordinates of point inside a tet
__device__
float4 InterpolateTet(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{

	float4 coord;

	float T = fabsf(dot(v1-v0,cross(v2-v0,v3-v0)));
	coord.w=__fdividef(fabsf(dot(v0-p,cross(v1-p,v2-p))),T);
	coord.z=__fdividef(fabsf(dot(v0-p,cross(v1-p,v3-p))),T);
	coord.y=__fdividef(fabsf(dot(v0-p,cross(v2-p,v3-p))),T);
	coord.x=__fdividef(fabsf(dot(v1-p,cross(v2-p,v3-p))),T);
	return coord;
}
// look up in 3D noise texture
__device__
float3 noise3D(float3 p)
{
    float4 n = tex3D(noiseTex, p.x, p.y, p.z);
    return make_float3(n.x, n.y, n.z);
}

__device__
float3 fractalSum3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += noise3D(p*freq)*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

__device__
float3 turbulence3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += fabs(noise3D(p*freq))*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

// integrate particle attributes
__global__ void
integrateD(float4* newPos, float4* newVel, int1 *newTetId,
           float4* oldPos, float4* oldVel, int1 *oldTetId, 		   
           float deltaTime,
		   int numParticles,
		   float particleLifetime)//, float4 *intersection
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;

	volatile float4 posData = oldPos[index];	// ensure coalesced reads
    volatile float4 velData = oldVel[index];
	volatile int1 tetIdData = oldTetId[index];
    float3 pos = make_float3(posData.x, posData.y, posData.z);
    float3 vel = make_float3(velData.x, velData.y, velData.z);
	int4 tet,neighbors;
	float4 v0_4,v1_4,v2_4,v3_4;
	float3 v[4];
	int1 tetId,tetId_bak;
	tetId.x= tetIdData.x;
	float age = posData.w;	
	//float lifetime = velData.w;
	float lifetime = particleLifetime;
    //reset the paricle if it enter the buliding or go out of the bounding box, or the active area, or its age reaches the lifetime
	if(tetId.x==-2)
	{
		float4 startPos=tex1Dfetch(start_pos_Tex,index);
		pos= make_float3(startPos.x, startPos.y, startPos.z);
		tetId=tex1Dfetch(start_tetId_Tex,index);
		tet=tex1Dfetch(air_tet_Tex,tetId.x);
		v0_4=tex1Dfetch(air_ver_Tex,tet.x);
		v[0]=make_float3(v0_4.x,v0_4.y,v0_4.z);
		v1_4=tex1Dfetch(air_ver_Tex,tet.y);
		v[1]=make_float3(v1_4.x,v1_4.y,v1_4.z);
		v2_4=tex1Dfetch(air_ver_Tex,tet.z);
		v[2]=make_float3(v2_4.x,v2_4.y,v2_4.z);
		v3_4=tex1Dfetch(air_ver_Tex,tet.w);
		v[3]=make_float3(v3_4.x,v3_4.y,v3_4.z);
		float4 barryCoord=InterpolateTet(pos,v[0],v[1],v[2],v[3]);
	
		float4 vel_4=barryCoord.x*tex1Dfetch(air_wind_Tex,tet.x)+barryCoord.y*tex1Dfetch(air_wind_Tex,tet.y)+barryCoord.z*tex1Dfetch(air_wind_Tex,tet.z)+barryCoord.w*tex1Dfetch(air_wind_Tex,tet.w);
		vel_4.w=lifetime;
		newPos[index] = startPos;//make_float4(pos, 0);
		newVel[index] = vel_4;
		newTetId[index]=tetId;
		return;
	}
	if(age>=lifetime||length(vel)<1e-6||pos.x<d_activeArea[0]||pos.x>d_activeArea[1]||pos.y<d_activeArea[2]||pos.y>d_activeArea[3]||pos.z<d_activeArea[4]||pos.z>d_activeArea[5])
	{
		float4 startPos=make_float4(posData.x, posData.y, posData.z,lifetime);
		float4 vel_4=make_float4(velData.x, velData.y, velData.z,lifetime);
		newPos[index] = startPos;//make_float4(pos, 0);
		newVel[index] = vel_4;
		tetId.x=-2;
		newTetId[index]=tetId;
		return;
	}
	
    // update particle age
	age += deltaTime;
	if (age > lifetime) {
	    age = lifetime;
    }

    // apply accelerations
   // vel += params.gravity * deltaTime;

    // apply procedural noise
    //float3 noise = noise3D(pos*params.noiseFreq + params.time*params.noiseSpeed);
    //vel += noise * params.noiseAmp;

	
	
	//ray object intersection
	float t=0;
	float t_bak=t;
	int outIndex;
	/////////////////////////////////////////////////////////////////////////////////////debug
	//float3 tpos,tvel;
	/////////////////////////////////////////////////////////////////////////////////////
 	while(t<deltaTime)
	{
		tetId_bak=tetId;
		tet=tex1Dfetch(air_tet_Tex,tetId.x);
		neighbors=tex1Dfetch(air_neighbor_Tex,tetId.x);
		v0_4=tex1Dfetch(air_ver_Tex,tet.x);
		v[0]=make_float3(v0_4.x,v0_4.y,v0_4.z);
		v1_4=tex1Dfetch(air_ver_Tex,tet.y);
		v[1]=make_float3(v1_4.x,v1_4.y,v1_4.z);
		v2_4=tex1Dfetch(air_ver_Tex,tet.z);
		v[2]=make_float3(v2_4.x,v2_4.y,v2_4.z);
		v3_4=tex1Dfetch(air_ver_Tex,tet.w);
		v[3]=make_float3(v3_4.x,v3_4.y,v3_4.z);
		//find out the face out
		if(RayTriangleIntersect(v[1],v[2],v[3],pos,vel,t,t_bak))
		{
			tetId.x=neighbors.x;
			//tpos.x=t;
			outIndex=0;
		}
		else
		{
			//tpos.x=t;
			if(RayTriangleIntersect(v[0],v[2],v[3],pos,vel,t,t_bak))
			{
				tetId.x=neighbors.y;
				//tpos.y=t;
				outIndex=1;
			}
			else
			{
				//tpos.y=t;
				if(RayTriangleIntersect(v[0],v[1],v[3],pos,vel,t,t_bak))
				{
					tetId.x=neighbors.z;
					//tpos.z=t;
					outIndex=2;
				}
				else
				{
					//tpos.z=t;
					if(RayTriangleIntersect(v[0],v[1],v[2],pos,vel,t,t_bak))
					{
						tetId.x=neighbors.w;
						//tvel.x=t;
						outIndex=3;
					} 
					else
					{
						//tvel.x=t;
			// 			tetId.x=-1;			
			// 			t=0;
			// 			outIndex=-1;
						vel=noise3D(pos);
						float4 barryCoord=InterpolateTet(pos,v[0],v[1],v[2],v[3]);
						//newPos[index] = make_float4(barryCoord.x,barryCoord.y,barryCoord.z,5);
						//newPos[index] = make_float4(tpos,5);
						//tvel.y=t_bak;
						//newVel[index] = make_float4(tvel, lifetime);
						//newVel[index] = make_float4(barryCoord.w,tvel.x,tvel.y, lifetime);
						newPos[index] = make_float4(pos, lifetime-1);
						newVel[index] = make_float4(vel, lifetime);
						newTetId[index]=tetId_bak;
						return;
					}
				}
			}
		}
		//if the particle enter the building, change its direction.
		//if(tetId.x<0)
		if(tetId.x==-1)
		{
			pos += vel * t_bak;
			newPos[index] = make_float4(pos, age);
			++outIndex;
			float3 v0=v[outIndex%4];
			++outIndex;
			float3 v1=v[outIndex%4];
			++outIndex;
			float3 v2=v[outIndex%4];
			float3 face_normal=normalize(cross(v1-v0,v2-v0));
			vel-=dot(face_normal,vel)*face_normal;
			if(length(vel)<1e-6)
				vel=noise3D(pos);
			else
				vel=normalize(vel);
			newVel[index] = make_float4(vel, lifetime);
			newTetId[index]=tetId_bak;
			return;
		}
		else if(tetId.x==-2)//go out of the bounding box
		{
			pos += vel * t;
			newPos[index] = make_float4(pos, lifetime);
			newVel[index] = make_float4(vel, lifetime);
			newTetId[index]=tetId;
			return;
		}
		t_bak=t;
	}
	//intersection[index]=make_float4(pos+vel*t,t);
    // new position = old position + velocity * deltaTime
    pos += vel * deltaTime;
	//pos += vel * t;
	
	float4 barryCoord=InterpolateTet(pos,v[0],v[1],v[2],v[3]);
	
	float4 vel_4=barryCoord.x*tex1Dfetch(air_wind_Tex,tet.x)+barryCoord.y*tex1Dfetch(air_wind_Tex,tet.y)+barryCoord.z*tex1Dfetch(air_wind_Tex,tet.z)+barryCoord.w*tex1Dfetch(air_wind_Tex,tet.w);
	vel_4.w=lifetime;
    //vel *= params.globalDamping;

    // store new position and velocity
    newPos[index] = make_float4(pos, age);
    newVel[index] = vel_4;
	newTetId[index]=tetId_bak;
	
}

// calculate sort depth for each particle
__global__ void calcDepthD(float4* pos, float* keys, uint *indices, float3 vector, int numParticles)
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;

	volatile float4 p = pos[index];
	float key = -dot(make_float3(p.x, p.y, p.z), vector);        // project onto sort vector
	
	keys[index] = key;
	indices[index] = index;
}

#endif
