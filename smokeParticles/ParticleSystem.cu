#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
 
 /*
	This file contains simple wrapper functions that call the CUDA kernels
 */

#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "particles_kernel.cu"
#include "ParticleSystem.cuh"

extern "C"
{

hipArray *noiseArray;

void initCuda(bool bUseGL)
{
	if (bUseGL) {
	    cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
	} else {
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	}
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}
void setActiveArea(float *area)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_activeArea), area, sizeof(float)*6) );
}
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(unsigned int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

inline float frand()
{
    return rand() / (float) RAND_MAX;
}

// create 3D texture containing random values
void createNoiseTexture(int w, int h, int d)
{
    hipExtent size = make_hipExtent(w, h, d);
    uint elements = (uint) size.width*size.height*size.depth;

    float *volumeData = (float *)malloc(elements*4*sizeof(float));
    float *ptr = volumeData;
    for(uint i=0; i<elements; i++) {
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
    }


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    cutilSafeCall( hipMalloc3DArray(&noiseArray, &channelDesc, size) );

    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr   = make_hipPitchedPtr((void*)volumeData, size.width*sizeof(float4), size.width, size.height);
    copyParams.dstArray = noiseArray;
    copyParams.extent   = size;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );

    free(volumeData);

    // set texture parameters
    noiseTex.normalized = true;                      // access with normalized texture coordinates
    noiseTex.filterMode = hipFilterModeLinear;      // linear interpolation
    noiseTex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    noiseTex.addressMode[1] = hipAddressModeWrap;
    noiseTex.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(noiseTex, noiseArray, channelDesc));
}
void bindAirData(float4 *verPtr,float4 *windPtr,int4 *tetPtr,int4 *neighborPtr,unsigned int numVer,unsigned int numTet)
{
	//hipChannelFormatDesc channelDescf4 = hipCreateChannelDesc<float4>();
	//hipChannelFormatDesc channelDescui4 = hipCreateChannelDesc<uint4>();
	cutilSafeCall(hipBindTexture(0, air_ver_Tex, verPtr,numVer*sizeof(float4))); //, &channelDescf4
	cutilSafeCall(hipBindTexture(0, air_wind_Tex, windPtr,numVer*sizeof(float4))); //,&channelDescf4,
	cutilSafeCall(hipBindTexture(0, air_tet_Tex, tetPtr,numTet*sizeof(uint4))); //,&channelDescui4
	cutilSafeCall(hipBindTexture(0, air_neighbor_Tex, neighborPtr,numTet*sizeof(uint4)));//,&channelDescui4
}
void createStartPos(unsigned int num)
{
	cutilSafeCall(hipMalloc((void **) &d_start_pos, num*sizeof(float4)));
	cutilSafeCall(hipMalloc((void **) &d_start_tetId, num*sizeof(int1)));
}
void freeStartPos()
{
	if (d_start_pos) {
        cutilSafeCall(hipFree(d_start_pos));
        d_start_pos = 0;
    }
	if (d_start_tetId) {
        cutilSafeCall(hipFree(d_start_tetId));
        d_start_tetId = 0;
    }
}
void bindStartPos(float4 *startPosPer,int1 *startTetId,unsigned int beginIndex,unsigned int num,unsigned int numParticles)
{
	cutilSafeCall(hipMemcpy((void *)&d_start_pos[beginIndex], (void *) &startPosPer[beginIndex], num*sizeof(float4), hipMemcpyDeviceToDevice));
	cutilSafeCall(hipMemcpy((void *)&d_start_tetId[beginIndex], (void *) &startTetId[beginIndex], num*sizeof(int1), hipMemcpyDeviceToDevice));
	cutilSafeCall(hipBindTexture(0, start_pos_Tex, d_start_pos,numParticles*sizeof(float4))); //, &channelDescf4
	cutilSafeCall(hipBindTexture(0, start_tetId_Tex, d_start_tetId,numParticles*sizeof(int1))); //, &channelDescf4
}
void unbindStartPos()
{
	cutilSafeCall(hipUnbindTexture(start_pos_Tex)) ;
	cutilSafeCall(hipUnbindTexture(start_tetId_Tex)) ;	
}
void unbindAirData()
{
	cutilSafeCall(hipUnbindTexture(air_ver_Tex)) ;
	cutilSafeCall(hipUnbindTexture(air_wind_Tex)) ;
	cutilSafeCall(hipUnbindTexture(air_tet_Tex)) ;
	cutilSafeCall(hipUnbindTexture(air_neighbor_Tex)) ;
}
void 
integrateSystem(float4 *oldPos, float4 *newPos,
				float4 *oldVel, float4 *newVel,
				int1 *oldTetId, int1 *newTetId,
                float deltaTime,
                unsigned int numParticles,
				float particleLifetime)//, float4 *intersection
{
    int numThreads;
	int numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    integrateD<<< numBlocks, numThreads >>>(newPos, newVel, newTetId,
                                            oldPos, oldVel,oldTetId,											
                                            deltaTime,
											numParticles,
											particleLifetime);//, intersection
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");
}


void 
calcDepth(float4*  pos, 
		  float*   keys,		// output
          uint*    indices,		// output 
          float3   sortVector,
          unsigned int      numParticles)
{
    int numThreads;
	int numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcDepthD<<< numBlocks, numThreads >>>(pos, keys, indices, sortVector, numParticles);
    cutilCheckMsg("calcDepthD execution failed");
}

}   // extern "C"
